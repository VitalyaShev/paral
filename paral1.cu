#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <hipcub/hipcub.hpp>


using namespace std;

__global__ void compute(double* Carr1, double* Carr, int n, double h){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	if((i > 0) && (i < n - 1) && (j > 0) && (j < n - 1))
        Carr1[i*n+j] = Carr[i*n+j] + 1./(double)(n*n*4) * ((Carr[(i - 1)*n+j] - 2 * Carr[i*n+j] + Carr[(i + 1)*n+j]) + (Carr[i*n + j - 1] - 2 * Carr[i*n+j] + Carr[i*n +j + 1])) / (h * h);
}

__global__ void Max_Reduction(double* Carr, double* Carr1, int n, double* BlockErr){
    typedef hipcub::BlockReduce<double, 16, hipcub::BLOCK_REDUCE_WARP_REDUCTIONS, 16> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
    double thread_data=0.0;

    int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
    if ((i > 0) && (i < n - 1) && (j > 0) && (j < n - 1))
        thread_data = Carr1[i*n + j] - Carr[i*n + j];
    double aggregate = BlockReduce(temp_storage).Reduce(thread_data, hipcub::Max());
    __syncthreads();
    if (threadIdx.x == 0 && threadIdx.y == 0)
        BlockErr[blockIdx.y*gridDim.x + blockIdx.x] = aggregate;
}


int main() {
	int iter;
	double acc = 0.000001; 
	double err = 1;
	int n = 1024; 
	int iters = 1000000; 
	double step = 10. / (double)(n);
	iter = 0;
	double arr[n][n], arr1[n][n];
	double h = 1.0 / (double)n;
	arr[0][0] = arr1[0][0] = 30;
	arr[n-1][n - 1] = arr1[n - 1][n - 1] = 20;
	arr[0][n - 1] = arr1[0][n - 1] = 20;
	arr[n - 1][0] = arr1[n - 1][0] = 10;
	for (int i = 1; i < n - 1; i++) {
		arr[0][i] = arr[0][i - 1] + step;
		arr[n - 1][i] = arr[n - 1][i - 1] + step;
	}
	for (int j = 1; j < n - 1; j++) {
		arr[j][0] = arr[j - 1][0] + step;
		arr[j][n - 1] = arr[j - 1][n - 1] + step;
	}
	for (int i = 1; i < n - 1; i++)
		for (int j = 1; j < n - 1; j++)
			arr[i][j] = 0;
            for (int i = 0; i < n; i++)
                for (int j = 0; j < n; j++)
            arr1[i][j] = arr[i][j];
    double* Carr;
    double* Carr1;
    hipMalloc(&Carr, n*n*sizeof(double));
	hipMalloc(&Carr1, n*n*sizeof(double));

    dim3 BS = dim3(16,16);
	dim3 GS = dim3(ceil(n/(float)BS.x),ceil(n/(float)BS.y));

    double* CBlockErr;
    hipMalloc(&CBlockErr, GS.x*GS.y*sizeof(double));
    double BlockErr[GS.x*GS.y];

    hipMemcpy(Carr, arr, n*n*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Carr1, arr, n*n*sizeof(double), hipMemcpyHostToDevice);
		while ((err > acc) && (iter < iters)) {
			iter+=2;
            compute<<<GS,BS>>>(Carr1, Carr, n, h);
            compute<<<GS,BS>>>(Carr, Carr1, n, h);		
			if ((iter % 100 == 0) || (iter == 2)){  
            err = 0;
        	Max_Reduction<<<GS,BS>>>(Carr1, Carr, n, CBlockErr);
            hipDeviceSynchronize();
            hipMemcpy(BlockErr, CBlockErr, GS.x*GS.y*sizeof(double), hipMemcpyDeviceToHost);
            for (int i = 0; i < GS.x; i++)
                for (int j = 0; j < GS.y; j++)
                    err = max(err, BlockErr[i*GS.x + j]);
            }
        }        
	cout << iter << ' ' << err;
    hipFree(Carr);
	hipFree(Carr1);
    hipFree(CBlockErr);
    hipDeviceReset();
	return 0;
}
